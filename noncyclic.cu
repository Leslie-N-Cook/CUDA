
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<unistd.h>
#include<sys/types.h>

# define SIZE 10240         //total number of threads
const int N = 1024;         //threads per block
const int blocksize = 2;    //solution a and b requires 2 blocks

 /*******************************************************************/
 __global__ 
 void nonCyclic(unsigned long int *a_d, unsigned long int *b_d, unsigned long int *c_d)
{
    //tid : index of the thread
    int tid = blockIdx.x * blockDim.x * (SIZE / blockDim.x / gridDim.x) + threadIdx.x;
    
    for (int j = 0; j< (SIZE / blockDim.x / gridDim.x); ++j) // (10240 / 1024 / 2) = 5
    {
        c_d[tid + (blockDim.x * j)] = a_d[tid + (blockDim.x * j)] * b_d[tid + (blockDim.x * j)];
    }
}

int main()
{

    unsigned long int *a_d, *b_d, *c_d; // device copies of a, b, c
    unsigned long int *a_h, *b_h, *c_h; // host copies of a, b, c

    //NOTE:This gets the amount of bytes needed for the array
    //sizeof(unsigned long int) in bytes times the size of the array
    const unsigned long int iSize = SIZE * sizeof(unsigned long int);

    /****************** solution #1 - Two blocks: NONcyclic ******************/
    //allocates the memory on the CPU side with the size 
    //computed above needs unsigned long int* to make sure that it is an array of unsigned long ints
    a_h = (unsigned long int*)malloc(iSize);
    b_h = (unsigned long int*)malloc(iSize);
    c_h = (unsigned long int*)malloc(iSize);

    //NOTE:This must be done BEFORE copping memory
    //loading the arrays
    for (int n = 0; n < SIZE; ++n)
    {
        a_h[n] = (2 * n);         //even numbers in array a
        b_h[n] = ((2 * n) + 1);   //odd numbers in array b
        c_h[n] = 0;             //array c initialized to 0
    }

    //allocates the memory on the GPU size void** and & is just needed
    hipMalloc((void**) &a_d, iSize);
    hipMalloc((void**) &b_d, iSize);
    hipMalloc((void**) &c_d, iSize);

    //copies the memory on the cpu side to the GPU 
    hipMemcpy(a_d, a_h, iSize, hipMemcpyHostToDevice);
 	hipMemcpy(b_d, b_h, iSize, hipMemcpyHostToDevice);

    //1-D grid with 2 blocks
	dim3 gridDim_non(blocksize ,1); 	

    //1-D block with 1024 threads per block 
	dim3 blockDim_non(N, 1);

    //calls the GPU functions with the perameters a_d, b_d, c_d
	nonCyclic<<<gridDim_non, blockDim_non>>>(a_d, b_d, c_d);

    //copies the GPU memory to the CPU 
    hipMemcpy(c_h, c_d, iSize, hipMemcpyDeviceToHost);

    //deallocate GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    //display the solution
    printf("\nSolution 1:\nTwo blocks and Non-cyclic (c[0], c[10239]) = (");
    printf("%d", c_h[0]);
    printf(", ");
    printf("%d", c_h[10239]);
    printf(")\n");

    //deallocate CPU memory
    free(a_h);
    free(b_h);
    free(c_h);
    
     /*******************************************************************/
    return 0;
}